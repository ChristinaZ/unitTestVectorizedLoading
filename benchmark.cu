#include "hip/hip_runtime.h"
#include <unistd.h>

#include <cmath>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <cassert>
#include <hiprand.h>
#include <random>
#include "hip/hip_runtime_api.h"
#include <cub/block/block_load.cuh>
#include <cub/util_type.cuh>

using namespace hipcub;

#define CURAND_CALL(x)                                                     \
    do                                                                     \
    {                                                                      \
        hiprandStatus ret = (x);                                            \
        if (ret != HIPRAND_STATUS_SUCCESS)                                  \
        {                                                                  \
            printf("cuRAND Error %d at %s:%d\n", ret, __FILE__, __LINE__); \
            exit(1);                                                       \
        }                                                                  \
    } while (0)

class CudaException : public std::runtime_error
{
public:
    explicit CudaException(const std::string &what) : runtime_error(what) {}
};

inline void cuda_check_(hipError_t val, const char *file, int line)
{
    if (val != hipSuccess)
    {
        throw CudaException(std::string(file) + ":" + std::to_string(line) + ": CUDA error " + std::to_string(val) + ": " + hipGetErrorString(val));
    }
}

inline void cuda_check_last_error_(const char *file, int line)
{
    hipDeviceSynchronize();
    hipError_t err = hipPeekAtLastError();
    cuda_check_(err, file, line);
}

#define CUDA_CHECK(val)                         \
    {                                           \
        cuda_check_((val), __FILE__, __LINE__); \
    }
#define CUDA_CHECK_LAST_ERROR()                     \
    {                                               \
        cuda_check_last_error_(__FILE__, __LINE__); \
    }

template <typename InputIterator,
          typename OutputIterator,
          int BLOCK_THREADS,
          int ITEMS_PER_THREAD>
struct DeviceExample
{
    using KeyInT = detail::value_t<InputIterator>;
    using BlockLoadInputT = BlockLoad<KeyInT, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_VECTORIZE>;
    static constexpr int TILE_ITEMS = BLOCK_THREADS * ITEMS_PER_THREAD;

    InputIterator in;
    OutputIterator out;
    int total_num;

    __device__ DeviceExample(InputIterator in, OutputIterator out, int total_num)
        : in(in), out(out), total_num(total_num)
    {
        assert(total_num == BLOCK_THREADS * ITEMS_PER_THREAD);
    }
    __device__ __forceinline__ void VectorizedProcess()
    {
        KeyInT thread_data[ITEMS_PER_THREAD];
        int tile_base = blockIdx.x * TILE_ITEMS;
        int offset = threadIdx.x * ITEMS_PER_THREAD + tile_base;

        __shared__ typename BlockLoadInputT::TempStorage temp_storage;
        BlockLoadInputT(temp_storage).Load(in + tile_base, thread_data);

        for (int i = 0; i < ITEMS_PER_THREAD; i++)
        {
            out[offset + i] = thread_data[i] * 2;
        }
    }
};

template <typename InputIterator,
          typename OutputIterator,
          int BLOCK_THREADS,
          int ITEMS_PER_THREAD>
__global__ void testBlockLoad(InputIterator in, OutputIterator out, int total_num)
{
    using DeviceExampleT = DeviceExample<InputIterator, OutputIterator, BLOCK_THREADS, ITEMS_PER_THREAD>;

    DeviceExampleT(in, out, total_num).VectorizedProcess();
}


template <typename T, 
          int BLOCK_THREADS,
          int ITEMS_PER_THREAD>
struct DeviceExampleTwo
{
    using BlockLoadInputT = BlockLoad<T, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_VECTORIZE>;
    static constexpr int TILE_ITEMS = BLOCK_THREADS * ITEMS_PER_THREAD;

    T* in;
    T* out;
    int total_num;

    __device__ DeviceExampleTwo(T* in, T* out, int total_num)
        : in(in), out(out), total_num(total_num)
    {
        assert(total_num == BLOCK_THREADS * ITEMS_PER_THREAD);
    }
    __device__ __forceinline__ void VectorizedProcess()
    {
        T thread_data[ITEMS_PER_THREAD];
        int tile_base = blockIdx.x * TILE_ITEMS;
        int offset = threadIdx.x * ITEMS_PER_THREAD + tile_base;

        __shared__ typename BlockLoadInputT::TempStorage temp_storage;
        BlockLoadInputT(temp_storage).Load(in + tile_base, thread_data);

        for (int i = 0; i < ITEMS_PER_THREAD; i++)
        {
            out[offset + i] = thread_data[i] * 2;
        }
    }
};

template <typename T,
          int BLOCK_THREADS,
          int ITEMS_PER_THREAD>
__global__ void testBlockLoadTwo(T* in, T* out, int total_num)
{
    using DeviceExampleT = DeviceExampleTwo<T, BLOCK_THREADS, ITEMS_PER_THREAD>;

    DeviceExampleT(in, out, total_num).VectorizedProcess();
}

template <typename T, 
          int BLOCK_THREADS,
          int ITEMS_PER_THREAD>
struct DeviceExampleConst
{
    using BlockLoadInputT = BlockLoad<T, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_VECTORIZE>;
    static constexpr int TILE_ITEMS = BLOCK_THREADS * ITEMS_PER_THREAD;

    const T* in;
    T* out;
    int total_num;

    __device__ DeviceExampleConst(const T* in, T* out, int total_num)
        : in(in), out(out), total_num(total_num)
    {
        assert(total_num == BLOCK_THREADS * ITEMS_PER_THREAD);
    }
    __device__ __forceinline__ void VectorizedProcess()
    {
        T thread_data[ITEMS_PER_THREAD];
        int tile_base = blockIdx.x * TILE_ITEMS;
        int offset = threadIdx.x * ITEMS_PER_THREAD + tile_base;

        __shared__ typename BlockLoadInputT::TempStorage temp_storage;
        BlockLoadInputT(temp_storage).Load(in + tile_base, thread_data);

        for (int i = 0; i < ITEMS_PER_THREAD; i++)
        {
            out[offset + i] = thread_data[i] * 2;
        }
    }
};

template <typename T,
          int BLOCK_THREADS,
          int ITEMS_PER_THREAD>
__global__ void testBlockLoadConst(const T* in, T* out, int total_num)
{
    using DeviceExampleT = DeviceExampleConst<T, BLOCK_THREADS, ITEMS_PER_THREAD>;

    DeviceExampleT(in, out, total_num).VectorizedProcess();
}

int main(int argc, char **argv)
{
    constexpr int BLOCK_NUM = 10;
    constexpr int BLOCK_THREADS = 256;
    constexpr int ITEMS_PER_THREAD = 4;

    int size = ITEMS_PER_THREAD * BLOCK_THREADS * BLOCK_NUM;
    int size_in_bytes = size * sizeof(float);

    using InputIterator =  float *;
    using OutputIterator = float *;

    InputIterator d_in = nullptr;
    OutputIterator d_out = nullptr;

    CUDA_CHECK(hipMalloc((void **)&d_in, size_in_bytes));
    CUDA_CHECK(hipMalloc((void **)&d_out, size_in_bytes));

    hiprandGenerator_t gen_;
    CURAND_CALL(hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen_, std::random_device{}()));
    CURAND_CALL(hiprandGenerateUniform(gen_, d_in, size));

    testBlockLoad<InputIterator, OutputIterator, BLOCK_THREADS, ITEMS_PER_THREAD><<<BLOCK_NUM, BLOCK_THREADS>>>(d_in, d_out, size);
    CUDA_CHECK_LAST_ERROR();

    testBlockLoadTwo<float, BLOCK_THREADS, ITEMS_PER_THREAD><<<BLOCK_NUM, BLOCK_THREADS>>>(d_in, d_out, size);
    CUDA_CHECK_LAST_ERROR();

    testBlockLoadConst<float, BLOCK_THREADS, ITEMS_PER_THREAD><<<BLOCK_NUM, BLOCK_THREADS>>>(d_in, d_out, size);
    CUDA_CHECK_LAST_ERROR();

    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
    CURAND_CALL(hiprandDestroyGenerator(gen_));
}
